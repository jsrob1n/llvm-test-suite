
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>

__global__ void kernel() {
  // Our reference output contains the line number of this assert() call; be
  // careful when modifying the parts of this file above this line.
  assert(false);
}

int main() {
  kernel<<<1,1>>>();
  hipError_t err = hipDeviceSynchronize();
  if (err != hipErrorAssert)
    return err;
  return 0;
}
